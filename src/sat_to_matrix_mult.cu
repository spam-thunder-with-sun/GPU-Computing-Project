#include "hip/hip_runtime.h"
using namespace std;

#include <vector>
#include <iostream>
#include <string>
#include <cmath>
#include <hip/hip_runtime.h>
#include "create_matrix.h"

int literals_ = 0;
int clauses_ = 0;

void print_solution(vector<bool> sol) 
{
    for (int i = 1; i <= sol.size() / 2; ++i)
        if (sol[i])
            cout << i << " ";
        else
            cout << "-" << i << " ";
    cout << endl;
}

bool sat(vector<vector<bool>> &M, vector<bool> &sol) 
{
    bool res;

    for (int i = 0; i < clauses_; ++i)
    {   
        res = false;

        for (int j = 1; j < literals_ * 2 + 1 && !res; ++j)
            res = M[i][j] && sol[j];

        if(!res)
            return false;
    }    

    return res;
}

__global__ void sat_kernel(vector<vector<bool>> &M, vector<bool> &sol, bool *res) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < clauses_) 
    {
        res[i] = false;
        for (int j = 1; j < literals_ * 2 + 1 && !res[i]; ++j)
            res[i] = M[i][j] && sol[j];
    }
}

void find_solution (vector<vector<bool>> &M) 
{
    vector<bool> vec (literals_ * 2 + 1, false);
    bool issat = false;

    for (int sol = 0; sol <= ((unsigned long long)1 << literals_) - 1; ++sol) 
    {

        for (int i = 0; i < literals_; ++i) 
        {
            vec[i + 1] = (sol >> i) & 1;
            vec[i + literals_ + 1] = !(vec[i + 1]);
        }

        if (sat(M, vec)) 
        {
            if(!issat)
            {
                issat = true;
                
            }
            cout << "SAT:";
            print_solution(vec);
        }
    }

    if(!issat)
        cout << "UNSAT!" << endl;
}

//Da testare
bool copy_to_gpu(vector<vector<int>> &matrix, int literals, int clauses)
{
    long nChunksEveryHalfClause = ceil(literals / sizeof(unsigned int));
    long nChunksEveryClause = nChunksEveryHalfClause * 2;
    size_t sizeMatrix = nChunksEveryClause * clauses * sizeof(unsigned int);

    unsigned int *matrix_h, *matrix_d;

    //Alloco il vettore lato host
    matrix_h = (unsigned int*)malloc(sizeMatrix);
    //Riempio il vettore lato host
    for(int i = 0; i < matrix.size(); ++i)
    {
        //Inizializzo a tutti 0
        for(int j = 0; j < nChunksEveryClause; ++j)
            matrix_h[nChunksEveryClause * i + j] = 0;

        //Setto a 1 i bit corrispondenti
        for(int j = 0; j < matrix[i].size(); ++j)
        {
            int chunk = matrix[i][j] / nChunksEveryHalfClause;
            int offset = matrix[i][j] % nChunksEveryHalfClause; //Likely uses the result of the division
            if(matrix[i][j] > 0)
                matrix_h[nChunksEveryClause * i + chunk] |= 1 << offset;
            else
                matrix_h[nChunksEveryClause * i + nChunksEveryHalfClause - chunk ] |= 1 << -offset;
        }
    }       

    //Alloco il vettore lato device copiandolo da quello lato host
    hipMalloc((void**)&matrix_d, size);
    hipMemcpy(matrix_d, matrix_h, size, hipMemcpyHostToDevice);

    return true;
}


int main() 
{
    //input/dimacs/jnh1.cnf
    //input/3sat/uf20-01.cnf
    //input/small.cnf
    //input/tutorial.cnf
    CreateMatrix *foo = new CreateMatrix("input/small.cnf", true);
    if (foo->get_error())  return(1);
    vector<vector<bool>> matrix = foo->get_matrix();
    literals_ = foo->get_literals();
    clauses_ = foo->get_clauses();

    copy_to_gpu(matrix, literals_, clauses_);
    find_solution(matrix);

    cout << "Fine" << endl;

    return 0;
}