#include "hip/hip_runtime.h"
using namespace std;

#include <vector>
#include <iostream>
#include <string>
#include <cmath>
#include <hip/hip_runtime.h>
#include "create_matrix.h"
#include <bitset>

#define mytype unsigned short

int literals_ = 0;
int clauses_ = 0;

void print_solution(vector<bool> sol) 
{
    for (int i = 1; i <= sol.size() / 2; ++i)
        if (sol[i])
            cout << i << " ";
        else
            cout << "-" << i << " ";
    cout << endl;
}

bool sat(vector<vector<bool>> &M, vector<bool> &sol) 
{
    bool res;

    for (int i = 0; i < clauses_; ++i)
    {   
        res = false;

        for (int j = 1; j < literals_ * 2 + 1 && !res; ++j)
            res = M[i][j] && sol[j];

        if(!res)
            return false;
    }    

    return res;
}

void find_solution (vector<vector<bool>> &M) 
{
    vector<bool> vec (literals_ * 2 + 1, false);
    bool issat = false;

    for (int sol = 0; sol <= ((unsigned long long)1 << literals_) - 1; ++sol) 
    {

        for (int i = 0; i < literals_; ++i) 
        {
            vec[i + 1] = (sol >> i) & 1;
            vec[i + literals_ + 1] = !(vec[i + 1]);
        }

        if (sat(M, vec)) 
        {
            if(!issat)
            {
                issat = true;
                
            }
            cout << "SAT:";
            print_solution(vec);
        }
    }

    if(!issat)
        cout << "UNSAT!" << endl;
}

//Testato
pair<mytype *, mytype *> copy_to_gpu(vector<vector<int>> &matrix, int literals, int clauses)
{
    if(matrix.empty())
        return make_pair(nullptr, nullptr);

    int nChunksEveryHalfClause = ceil((float)literals / (sizeof(mytype) * 8));
    int nChunksEveryClause = nChunksEveryHalfClause * 2;
    size_t sizeMatrix = nChunksEveryClause * clauses * (sizeof(mytype));
    mytype *matrix_h, *matrix_d;

    //Alloco il vettore lato host
    matrix_h = (mytype*)malloc(sizeMatrix);
    //Riempio il vettore lato host
    for(int i = 0; i < matrix.size(); ++i)
    {
        //Inizializzo a tutti 0
        for(int j = 0; j < nChunksEveryClause; ++j)
            matrix_h[nChunksEveryClause * i + j] = 0;
        //Setto a 1 i bit corrispondenti
        for(int j = 0; j < matrix[i].size(); ++j)
        {
            int _value = abs(matrix[i][j]);
            int offset = (int)(_value-1) % (int)(sizeof(mytype) * 8);
            int chunk = nChunksEveryHalfClause - 1 - ((int)(_value-1) / (int)(sizeof(mytype) * 8));
            if(matrix[i][j] > 0)
                matrix_h[nChunksEveryClause * i + chunk] |= 1 << offset;
            else if(matrix[i][j] < 0)
                matrix_h[nChunksEveryClause * i + nChunksEveryHalfClause + chunk ] |= 1 << offset;
        }
    }  

    //Alloco il vettore lato device copiandolo da quello lato host
    hipMalloc((void**)&matrix_d, sizeMatrix);
    hipMemcpy(matrix_d, matrix_h, sizeMatrix, hipMemcpyHostToDevice);

    return make_pair(matrix_h, matrix_d);
}

//Testato
void printCompressMatrix(mytype *matrix, int literals, int clauses)
{
    int nChunksEveryHalfClause = ceil((float)literals / (sizeof(mytype) * 8));
    int nChunksEveryClause = nChunksEveryHalfClause * 2;

    cout << "Print compress matrix: " << endl;
    cout << "nChunksEveryHalfClause: " << nChunksEveryHalfClause << endl;
    cout << "nChunksEveryClause: " << nChunksEveryClause << endl;
    cout << "literals: " << literals << endl;
    cout << "clauses: " << clauses << endl;
    cout << "sizeof(mytype): " << sizeof(mytype) << endl;
    cout << endl;

    for(int i = 0; i < clauses; ++i)
    {
        for(int j = 0; j < nChunksEveryClause; ++j)
        {
            bitset<sizeof(mytype) * 8> x(matrix[nChunksEveryClause * i + j]);
            cout << x << " ";
            //cout << x << "(" << matrix[nChunksEveryClause * i + j] << ") ";
        }
        cout << endl;
    }
    cout << endl;
}

int main() 
{
    cout << "-----------------------------------------------" << endl;
    cout << endl << "*** Start ***" << endl << endl;
    //input/dimacs/jnh1.cnf
    //input/3sat/uf20-01.cnf
    //input/small.cnf
    //input/tutorial.cnf
    //input/hole6.cnf
    CreateMatrix *matrix = new CreateMatrix("input/small.cnf", true);
    if (matrix->get_error())  return(1);
    vector<vector<bool>> bool_matrix = matrix->get_boolean_matrix();
    vector<vector<int>> int_matrix = matrix->get_int_matrix();
    literals_ = matrix->get_literals();
    clauses_ = matrix->get_clauses();

    //find_solution(bool_matrix);
    //cout << endl;

    cout << "Copy to gpu " <<  endl;
    pair<mytype *, mytype *> ref = copy_to_gpu(int_matrix, literals_, clauses_);
    printCompressMatrix(ref.first, literals_, clauses_);

    cout << endl << "*** End ***" << endl;
    cout << "-----------------------------------------------" << endl;

    return 0;
}

/*
I dati sono salvati in formato big endian, quindi il bit più significativo è il primo
*/
